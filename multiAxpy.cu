#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <time.h>

__global__ void axpy(float a, float *xVec, float *yVec){
	//block.Idx.x, threadIdx.x, blockDim.x
	int subID;
	for(subID=0; subID < 8; subID++){
		int idx = subID +(threadIdx.x*8) + blockIdx.x*(blockDim.x*8);
		yVec[idx] = a*xVec[idx] + yVec[idx];
	}
}

int main(int argc, char** argv){

	int N = atoi(argv[1]);
	float a = 0.5;

	float *x_host = (float *)malloc(N*sizeof(float));
	float *y_host = (float *)malloc(N*sizeof(float));

	int i;
	for(i=0; i<N; i++){
		x_host[i] = (float)i;
		y_host[i] = 1.0;
	}

	float *z = (float *)malloc(N*sizeof(float));
	clock_t startTime = clock();
	int rep;
	int nReps = 100;
	for(rep=0; rep<nReps; rep++){
		for(i=0; i<N; i++) z[i] = a*x_host[i] + y_host[i];
	}
	clock_t endTime = clock();
	float cpuTime = float(endTime-startTime)/(float)(nReps*CLOCKS_PER_SEC);
	printf("CPU took %e seconds to run %d entry axpy \n", cpuTime, N);
	free(z);
	z = NULL;


	if(N<40){
		float* x_device;
		float* y_device;
		hipMalloc((void**)&x_device, N*sizeof(float));
		hipMalloc((void**)&y_device, N*sizeof(float));

		// copy from host to device
		hipMemcpy(x_device, x_host, N*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(y_device, y_host, N*sizeof(float), hipMemcpyHostToDevice);  

		int n_entries_per_thread = 8;
		int n_threads_per_block = 16/n_entries_per_thread;
		int n_blocks = N/(n_threads_per_block*n_entries_per_thread);

		axpy <<<n_blocks, n_threads_per_block>>>  (a, x_device, y_device);

	
		hipMemcpy(y_host, y_device, N*sizeof(float), hipMemcpyDeviceToHost);
	
		int j;
		printf("For small case, y at end is: \n");
		for(j=0; j<N; j++) printf("%f \n", y_host[j]);
				

		hipFree(x_device);
		hipFree(y_device);
		x_device = NULL;
		y_device = NULL;
	}
	
	//starting GPU time
	startTime = clock();
	
	
	float* x_device;
	float* y_device;
	hipMalloc((void**)&x_device, N*sizeof(float));
	hipMalloc((void**)&y_device, N*sizeof(float));

	// copy from host to device
	hipMemcpy(x_device, x_host, N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(y_device, y_host, N*sizeof(float), hipMemcpyHostToDevice);  

	int n_entries_per_thread = 8;
	int n_threads_per_block = 16/n_entries_per_thread;
	int n_blocks = N/(n_threads_per_block*n_entries_per_thread);

	axpy <<<n_blocks, n_threads_per_block>>>  (a, x_device, y_device);

	
	hipMemcpy(y_host, y_device, N*sizeof(float), hipMemcpyDeviceToHost);
	
	hipFree(x_device);
	hipFree(y_device);
	x_device = NULL;
	y_device = NULL;
	
	endTime = clock();
	float timeGPU = (float)(endTime-startTime)/(float)(nReps*CLOCKS_PER_SEC);
	printf("Time to run GPU code: %e \n", timeGPU);	

	free(x_host);
	free(y_host);
	x_host=NULL;
	y_host=NULL;
	
	return 0;
}
