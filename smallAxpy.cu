#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

__global__ void axpy(float a, float *xVec, float *yVec){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	yVec[idx] = a*xVec[idx] + yVec[idx];
}

int main(int argc, char** argv){

	int N = atoi(argv[1]);
	float a = 0.5;

	float *x_host = (float *)malloc(N*sizeof(float));
	float *y_host = (float *)malloc(N*sizeof(float));

	int i;
	for(i=0; i<N; i++){
		x_host[i] = (float)i;
		y_host[i] = 1.0;
	}

	for(i=0; i<N; i++) printf("i: %d, x[i]:, %f, y[i]: %f \n", i, x_host[i], y_host[i]);

	float * x_device;
	float * y_device;
	hipMalloc((void**)&x_device, N*sizeof(float));
	hipMalloc((void**)&y_device, N*sizeof(float));

	// copy from host to device
	hipMemcpy(x_device, x_host, N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(y_device, y_host, N*sizeof(float), hipMemcpyHostToDevice);  

	int n_threads_per_block = 16;
	int n_blocks = N/n_threads_per_block;

	axpy <<<n_blocks, n_threads_per_block>>>  (a, x_device, y_device);

	
	hipMemcpy(y_host, y_device, N*sizeof(float), hipMemcpyDeviceToHost);
	
	hipFree(x_device);
	hipFree(y_device);
	x_device = NULL;
	y_device = NULL;

	printf("After axpy (0.5*x + y): \n");
	for(i=0; i<N; i++) printf("%f \n", y_host[i]);

	free(x_host);
	free(y_host);
	x_host=NULL;
	y_host=NULL;

	return 0;
}
