#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <time.h>

__global__ void axpy(float a, float *xVec, float *yVec){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	yVec[idx] = a*xVec[idx] + yVec[idx];
}

int main(int argc, char** argv){

	int N = atoi(argv[1]);
	float a = 0.5;

	float *x_host = (float *)malloc(N*sizeof(float));
	float *y_host = (float *)malloc(N*sizeof(float));

	int i;
	for(i=0; i<N; i++){
		x_host[i] = (float)i;
		y_host[i] = 1.0;
	}

	float *z = (float *)malloc(N*sizeof(float));
	clock_t startTime = clock();
	int rep;
	int nReps = 100;
	for(rep=0; rep<nReps; rep++){
		for(i=0; i<N; i++) z[i] = a*x_host[i] + y_host[i];
	}
	clock_t endTime = clock();
	float cpuTime = float(endTime-startTime)/(float)(nReps*CLOCKS_PER_SEC);
	printf("CPU took %e seconds to run %d entry axpy \n", cpuTime, N);
	free(z);
	z = NULL;

	//starting GPU time
	startTime = clock();
	
	
	float* x_device;
	float* y_device;
	hipMalloc((void**)&x_device, N*sizeof(float));
	hipMalloc((void**)&y_device, N*sizeof(float));

	// copy from host to device
	hipMemcpy(x_device, x_host, N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(y_device, y_host, N*sizeof(float), hipMemcpyHostToDevice);  

	int n_threads_per_block = 16;
	int n_blocks = N/n_threads_per_block;

	axpy <<<n_blocks, n_threads_per_block>>>  (a, x_device, y_device);

	
	hipMemcpy(y_host, y_device, N*sizeof(float), hipMemcpyDeviceToHost);
	
	hipFree(x_device);
	hipFree(y_device);
	x_device = NULL;
	y_device = NULL;
	
	endTime = clock();
	float timeGPU = (float)(endTime-startTime)/(float)(nReps*CLOCKS_PER_SEC);
	printf("Time to run GPU code: %e \n", timeGPU);	

	free(x_host);
	free(y_host);
	x_host=NULL;
	y_host=NULL;

	return 0;
}
